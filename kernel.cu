#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdio.h>
#include "helper_timer.h"

void CleanUp();


__global__ void ColorInvert(float *d_in,float *d_out)
{
	// variable declarations
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	// code
	if (row < 6162980 && (row % 3 == 0))
	{
		const float red = d_in[row];
		const float green = d_in[row + 1];
		const float blue = d_in[row + 2];

			d_out[row]   = 255.0f-red;
			d_out[row+1] = 255.0f-green;
			d_out[row+2] = 255.0f-blue;

	}

}


float *d_in=NULL;
float *d_out=NULL;

void ProcessImage(float *rgb,float *x1,float *timeonGpu, char deviceName[])
{

	hipError_t err;

	hipDeviceProp_t dev_prop;
	hipGetDeviceProperties(&dev_prop, 0);


	err = hipMalloc((void **)&d_in, sizeof(float) * 6162980);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n", hipGetErrorString(err), __FILE__, __LINE__);

		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_out, sizeof(float) * 6162980);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n", hipGetErrorString(err), __FILE__, __LINE__);

		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_out, x1, sizeof(float) * 6162980, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

    //MessageBox(NULL,TEXT("After Mem Alloc"),TEXT("Here"),MB_OK);
	err = hipMemcpy(d_in, rgb, sizeof(float) * 6162980, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	sprintf(deviceName, "%s",dev_prop.name);

	dim3 DimGrid = dim3(ceil((double) 6162980/1024),1, 1);
	dim3 DimBlock = dim3(1024, 1, 1);

	// start timer
    StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	ColorInvert<<<DimGrid, DimBlock>>>(d_in,d_out);

	sdkStopTimer(&timer);
	*timeonGpu = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	err = hipMemcpy(x1, d_out, sizeof(float) * 6162980, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		printf("GPU Memory Fatal Error = %s In File Name %s At Line No. %d.\nExitting ...\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	CleanUp();

}


void CleanUp()
{
	if (d_in)
	{
		hipFree(d_in);
		d_in = NULL;
	}


	if (d_out)
	{
		hipFree(d_out);
		d_out = NULL;
	}


}